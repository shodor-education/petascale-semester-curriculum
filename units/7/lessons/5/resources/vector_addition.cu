#include "hip/hip_runtime.h"
#include<stdio.h>


#define SIZE 1000000
/*
 * Function: add_host
 * ----------------------------
 *   Serially adds the values in vector a and b to sum
 *
 *   a: vector a
 *   b: vector b
 *   n: size of the vectors 
 *   sum: vector to store results
 */
void add_host(int* a, int* b, int* sum, int n) {
	for (int i = 0; i < n; i++)
	{
		sum[i] = a[i] + b[i];
	}
}
/*
 * Kernel - Add vectors 
 * ----------------------------
 *   Each thread adds the values from vector a and b to sum
 *	 corresponding to the thread index
 *
 *   a: vector a
 *   b: vector b
 *   sum: vector to store results
 */
__global__ void add_device(int* a, int* b, int* sum, int n) {
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_id < n)
		sum[thread_id] = a[thread_id] + b[thread_id];
}
/*
 * Function: sum_vect
 * ----------------------------
 *   Adds and prints all the elements in vector vect for validation
 *
 *   vect: vector
 */
void sum_vect(int* vect)
{
	int total = 0;
	//sum all the elements 
	for (int i = 0; i < SIZE; i++) 
	{ 
		total += vect[i];
	}
	//print results
	printf("%d \n", total);
}

/*********************************************
 * main
 *********************************************/
int main(void) {
	//host vectors 
	int *h_a, *h_b, *h_sum;
	//device vectors
	int* d_a, * d_b, * d_sum;
	size_t size_vect = SIZE*sizeof(int); /* size of the total vectors necessary to allocate memory */
	
	//allocate memory for the vectors on host (cpu)
	h_a = (int*)malloc(size_vect);
	h_b = (int*)malloc(size_vect);
	h_sum = (int*)malloc(size_vect);

	//allocate memory for the vectors on device (gpu)
	hipMalloc((void **)&d_a, size_vect);
	hipMalloc((void **)&d_b, size_vect);
	hipMalloc((void **)&d_sum, size_vect);

	//initialize the vectors each with value 1
	for (int i = 0; i < SIZE; i++) {
		h_a[i] = 1;
		h_b[i] = 1;
	}

	//use serial function for vector addition
	add_host(h_a, h_b, h_sum,SIZE);
	//Verify the result by adding all the sum, should be 2 * SIZE
	printf("Host sum:\n");
	sum_vect(h_sum);
	
	//Start CUDA processing
	// Copy vector host values to device
	hipMemcpy(d_a, h_a, size_vect, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size_vect, hipMemcpyHostToDevice);

	//define number of threads
	int threads = 1024;
	//define block size in integer
	int block_size = (int)ceil((float)SIZE / threads);
	//execute the kernel with block size and number of threads
	add_device << <block_size, threads >>> (d_a, d_b, d_sum, SIZE);

	// Copy result back to host
	hipMemcpy(h_sum, d_sum, size_vect, hipMemcpyDeviceToHost);
	
	//Verify the result by adding all the sum, should be 2 * SIZE
	printf("Device sum:\n");
	sum_vect(h_sum);
	
	// Release all device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_sum);

	// Release all host memory
	free(h_a);
	free(h_b);
	free(h_sum);
}