#include "hip/hip_runtime.h"
/* Blue Waters Petascale Semester Curriculum v1.0
 * Unit 7: CUDA
 * Lesson 6: CUDA Atomic Functions
 * File: atomic_addition.cu
 * Developed by Sanish Rai for the Shodor Education Foundation, Inc.
 *
 * Copyright (c) 2020 The Shodor Education Foundation, Inc.
 *
 * Browse and search the full curriculum at
 * <http://shodor.org/petascale/materials/semester-curriculum>.
 *
 * We welcome your improvements! You can submit your proposed changes to this
 * material and the rest of the curriculum in our GitHub repository at
 * <https://github.com/shodor-education/petascale-semester-curriculum>.
 *
 * We want to hear from you! Please let us know your experiences using this
 * material by sending email to petascale@shodor.org
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Affero General Public License as published
 * by the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU Affero General Public License for more details.
 *
 * You should have received a copy of the GNU Affero General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

// INSTRUCTIONS HOW TO RUN
//
// Replace program_name with the name of the .cu file
// Compile with: nvcc program_name.cu -o program_name
// Run with    : ./program_name

#include<stdio.h>


#define SIZE 900000

/*
 * Kernel - Count threads without atomic
 * ----------------------------
 *   Each thread adds its value to sum
 *
 *   a: vector a
 *   sum: to store results
 *   n: maximum size of vector a
 */
__global__ void simple_count(int *a, int *sum, int n) {
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_id < n)
		*sum =*sum+a[thread_id];
}

/*
 * Kernel - Count threads with atomic
 * ----------------------------
 *   Each thread it's values sum using atomicAdd
 *
 *   a: vector a
 *   sum:  to store results
 *   n: maximum size of vector a

 */
__global__ void atomic_count(int *a, int *sum, int n) {
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_id < n)
		atomicAdd(sum,a[thread_id]);
}


/*********************************************
 * main
 *********************************************/
int main(void) {
	//host variables 
	int *h_var, *h_sum ;
	int sum=0;

	//device variables
	int* d_var, *d_sum;
	
	size_t size_vect = SIZE*sizeof(int); /* size of the total vectors necessary to allocate memory */
	
	//allocate memory for the variables on host (cpu)
	h_var = (int*)malloc(size_vect);
	h_sum = (int*)malloc(sizeof(int));
	h_sum=&sum;/* h_sum is to store the sum on the host device */
	
	//allocate memory for the variables on device (gpu)
	hipMalloc((void **)&d_var, size_vect);
	hipMalloc((void **)&d_sum, size_vect);
	hipMemset ((void **)d_sum,0, sizeof(int));
	
	//initialize the vectors each with value 1
	for (int i = 0; i < SIZE; i++) {
		h_var[i] = 1;
	}

	//Start CUDA processing
	// Copy host values to device
	hipMemcpy(d_var, h_var, size_vect, hipMemcpyHostToDevice);

	//define number of threads
	int threads = 1024;
	//define block size in integer
	int block_size = (int)ceil((float)SIZE / threads);
	
	//execute the kernel with block size and number of threads
	simple_count << <block_size, threads >>> (d_var, d_sum, SIZE);

	// Copy result back to host
	hipMemcpy(h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
	
	//Verify the result, should be equal to SIZE
	printf("Result without atomic add : %d\n",sum);

	//reset d_sum to 0
	hipMemset ((void **)d_sum,0, sizeof(int));
	
	//execute the kernel with block size and number of threads
	atomic_count << <block_size, threads >>> (d_var, d_sum, SIZE);

	// Copy result back to host
	hipMemcpy(h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
	
	//Verify the result, should be equal to SIZE
	printf("Result using atomic add : %d\n",sum);
	
	// Release all device memory
	hipFree(d_var);

	// Release all host memory
	free(h_var);
}
