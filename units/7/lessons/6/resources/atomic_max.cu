#include "hip/hip_runtime.h"
// INSTRUCTIONS HOW TO RUN
//
// Replace program_name with the name of the .cu file
// Compile with: nvcc program_name.cu -o program_name
// Run with    : ./program_name

#include<stdio.h>


#define SIZE 900000

/*
 * Kernel - Find max without atomic
 * ----------------------------
 *   Each thread checks if max is greater than its value,
 *   if yes then replaces the max
 *
 *   a: vector a
 *   max: to store results
 *   n: maximum size of vector a
 */
__global__ void simple_max(int *a, int *max, int n) {
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_id < n && a[thread_id]>*max)
		*max =a[thread_id];
}

/*
 * Kernel - Find max with atomic
 * ----------------------------
 *   Each thread checks if max is greater than its value,
 *   if yes then replaces the max using atomicMax
 *
 *   a: vector a
 *   max:  to store results
 *   n: maximum size of vector a

 */
__global__ void atomic_max(int *a, int *max, int n) {
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_id < n)
		atomicMax(max,a[thread_id]);
}


/*********************************************
 * main
 *********************************************/
int main(void) {
	//host variables 
	int *h_var, *h_max ;
	int max=0;

	//device variables
	int* d_var, *d_max;
	
	size_t size_vect = SIZE*sizeof(int); /* size of the total vectors necessary to allocate memory */
	
	//allocate memory for the variables on host (cpu)
	h_var = (int*)malloc(size_vect);
	h_max = (int*)malloc(sizeof(int));
	h_max=&max;/* h_max is to store the max on the host device */
	
	//allocate memory for the variables on device (gpu)
	hipMalloc((void **)&d_var, size_vect);
	hipMalloc((void **)&d_max, size_vect);
	hipMemset ((void **)d_max,0, sizeof(int));
	
	//initialize the vectors each with value 1
	for (int i = 0; i < SIZE; i++) {
		h_var[i] = i+1;
	}

	//Start CUDA processing
	// Copy host values to device
	hipMemcpy(d_var, h_var, size_vect, hipMemcpyHostToDevice);

	//define number of threads
	int threads = 1024;
	//define block size in integer
	int block_size = (int)ceil((float)SIZE / threads);
	
	//execute the kernel with block size and number of threads
	simple_max << <block_size, threads >>> (d_var, d_max, SIZE);

	// Copy result back to host
	hipMemcpy(h_max, d_max, sizeof(int), hipMemcpyDeviceToHost);
	
	//Verify the result, should be equal to SIZE
	printf("Result without atomic add : %d\n",max);

	//reset d_max to 0
	hipMemset ((void **)d_max,0, sizeof(int));
	
	//execute the kernel with block size and number of threads
	atomic_max << <block_size, threads >>> (d_var, d_max, SIZE);

	// Copy result back to host
	hipMemcpy(h_max, d_max, sizeof(int), hipMemcpyDeviceToHost);
	
	//Verify the result, should be equal to SIZE
	printf("Result using atomic add : %d\n",max);
	
	// Release all device memory
	hipFree(d_var);

	// Release all host memory
	free(h_var);
}